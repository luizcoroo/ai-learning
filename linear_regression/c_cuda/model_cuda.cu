
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>

__global__ void kernel_forward(const float *x, const float *w, float *y,
                               int n_rows, int n_columns) {
  int i = threadIdx.x;
  float sum = 0;
  if (i < n_rows)
    for (int j = 0; j < n_columns; j++)
      sum += x[i * n_columns + j] * w[j];

  y[i] = sum;
}

extern "C" void model_cuda_forward(const float *x, const float *w, float *y,
                                   int n_rows, int n_columns) {

  unsigned long x_size = sizeof(float) * n_rows * n_columns;
  unsigned long w_size = sizeof(float) * n_columns;
  unsigned long y_size = sizeof(float) * n_rows;

  float *d_x, *d_w, *d_y;
  hipMalloc((void **)&d_x, x_size);
  hipMalloc((void **)&d_w, w_size);
  hipMalloc((void **)&d_y, y_size);

  hipMemcpy(d_x, x, x_size, hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, w_size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, y_size, hipMemcpyHostToDevice);

  kernel_forward<<<1, n_rows>>>(d_x, d_w, d_y, n_rows, n_columns);
  hipDeviceSynchronize();

  hipMemcpy(y, d_y, y_size, hipMemcpyDeviceToHost);
}
